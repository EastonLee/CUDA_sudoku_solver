/*
 ============================================================================
 Name        : c_sudoku.c
 Author      : Easton Lee
 Version     :
 Copyright   : Your copyright notice
 Description : CUDA_sudoku_solver
 ============================================================================
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <sys/time.h>
#define ROWS 25
#define COLS ROWS
#define ROOT 5
#define NO_SOLUTION 0
#define MAY_HAVE_SOLUTION 1
#define GOT_SOLUTION 2
#define NO_CHANGE_SO_PAUSE -1
//typedef (long long) EASTON_TYPE;

#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

#define Runtime(...) do{\
	struct timeval start, end;\
	gettimeofday(&start, NULL);\
	__VA_ARGS__;\
	gettimeofday(&end, NULL);\
	printf(": Used time %f\n",(double) (end.tv_usec - start.tv_usec) / 1000000 + (double)(end.tv_sec-start.tv_sec));\
}while(0)

__device__ bool *d_B_change_occur , *d_B_no_solution , *d_B_got_solution ;
 bool h_B_change_occur = 1, h_B_no_solution = 0, h_B_got_solution = 0;

long long original_matrix[ROWS][COLS] = { 0 };
//{ROOT,0,7,9,0,0,0,2,0,4,0,0,0,0,0,0,0,0,0,0,0,3,6,0,1,0,5,0,8,0,0,0,0,2,1,7,0,0,0,4,0,0,0,0,0,0,0,0,0,0,9,0,5,0,0,0,8,0,0,3,0,0,9,0,0,0,0,7,2,0,0,0,0,0,3,0,0,0,6,0,0};
long long matrix[ROWS][COLS] ={0};
//{0,0,0,0,0,0,0,8,0,0,9,0,0,0,0,0,0,0,7,0,8,9,0,0,0,4,0,0,0,4,0,0,0,0,7,0,0,0,0,0,6,0,2,0,0,8,0,0,0,0,2,3,9,0,2,0,0,3,0,0,0,0,0,3,0,0,5,0,0,6,0,7,0,4,0,6,0,1,5,0,0};
//{ 0, 0, 4, 9, 5, 0, 0, 0, 0, 0, 0, 0, 2, 0, 0, 0, 3,	0, 0, 1, 0, 0, 0, 0, 9, 0, 8, 1, 9, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,	0, 5, 7, 0, 0, 6, 3, 0, 0, 0, 0, 0, 0, 0, 0, 0, 4, 0, 0, 0, 0, 0, 0, 8,	0, 5, 0, 0, 1, 0, 9, 5, 0, 9, 0, 0, 0, 4, 8, 0 };
//{1,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
//{0,0,4,9,5,0,0,0,0,0,0,0,2,0,0,0,3,0,0,1,0,0,0,0,9,0,8,1,9,0,0,0,0,0,0,0,0,0,0,0,0,0,5,7,0,0,6,3,0,0,0,0,0,0,0,0,0,4,0,0,0,0,0,0,8,0,5,0,0,1,0,9,5,0,9,0,0,0,4,8,0};
//{3,0,7,9,0,0,0,2,0,4,0,0,0,0,0,0,0,0,0,0,0,3,6,0,1,0,5,0,8,0,0,0,0,2,1,7,0,0,0,4,0,0,0,0,0,0,0,0,0,0,9,0,5,0,0,0,8,0,0,3,0,0,9,0,0,0,0,7,2,0,0,0,0,0,3,0,0,0,6,0,0};
__device__ long long (*d_matrix)[ROWS][COLS];//donnot read this shit-->//donot treat as a pointer, but as a matrix

int tree_length = 0, divide_times = 0, tree_change_times=0;
__device__ int *d_conflict_pos , *d_exhaust_pos;
int h_conflict_pos = -1, h_exhaust_pos = -1;


int simple_node_link[ROWS * COLS * 2]  ;

typedef struct node {
	long long (*current_matrix)[ROWS][COLS];
	int divide_from_x, divide_from_y;
	long long current_candicate;
	struct node * p_prev_node;
} node;

//node * head_node;
//node * tree;

void print_simple_node_link(int simple_node_link[ROWS * COLS * 2])
{
	int i;
	printf("\nnow the simple node link is like this\n");
	for(i=0;simple_node_link[i]!=-1;i++)
	{
		if(i%2 == 0)
			printf("(%d ",simple_node_link[i]);
		else
			printf("%d) ",simple_node_link[i]);
	}
	printf("\n\n");
}
__host__ __device__ int bit_count(long long foo) {
	long long i, sum = 0, tmp;
	for (i = 0; i < ROWS; i++) {
		tmp = 1 << i;
		sum += ((foo & tmp) >> i);
	}
	return sum;
}

__host__ __device__ int highest_bit(long long foo) {
	int i, tmp;
	for (i = ROWS-1; i >= 0; i--) {
		tmp = 1 << i;
		if ((foo & tmp) >> i)
			return i + 1;
	}
	return 0;
}

int calc_least_candicate(long long (*foo)[ROWS][COLS], int *least_x, int *least_y,
		int *least_candicate) {
	int i, tmp_x = -1, tmp_y = -1, tmp_least = 0xFF;
	for (i = 0; i < ROWS * COLS; i++) {
		if (((*foo)[i / ROWS][i % COLS]) & 0x8000000000000000) {
			long long tmp = (*foo)[i / ROWS][i % COLS];
			int bit_num = bit_count(tmp);
			if (bit_num < tmp_least) {
				tmp_x = i / COLS;
				tmp_y = i % COLS;
				tmp_least = bit_count((*foo)[tmp_x][tmp_y]);
			}
		}

	}

	*least_x = tmp_x;
	*least_y = tmp_y;
	*least_candicate = tmp_least;
	return tmp_least;
}

void print_matrix(long long foo[ROWS][COLS]) {
	int i;

	printf("\nThe current matrix is like this\n");
	for (i = 0; i < ROWS * COLS; i++) {
		if (!(i % COLS)) {
			printf("\n");
		}
		if ((foo[i / COLS][i % COLS]) & 0x8000000000000000)
			printf("_ ");
		else
			printf("%d ", foo[i / COLS][i % COLS]);

	}
	printf("\n");
}

void print_candicate_num(long long foo[ROWS][COLS]) {
	int i;

	printf("\nThe number of potential solutions is like this");
	for (i = 0; i < ROWS * COLS; i++) {
		if (!(i % COLS)) {
			printf("\n");
		}
		if ((foo[i / COLS][i % COLS]) & 0x8000000000000000)
			printf("%d ", bit_count(foo[i / COLS][i % COLS]));
		else
			printf("_ ");

	}
	printf("\n");
}

__global__ void VecAdd(const float* A, const float* B, float* C, int N) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < N)
		C[i] = A[i] + B[i];
}

__global__ void kernel_row_check(long long (*d_matrix)[ROWS][COLS],bool *d_B_change_occur,bool *d_B_no_solution,bool *d_B_got_solution,int *d_conflict_pos,int *d_exhaust_pos) {
	int i, bit_map = 0;

	*d_B_change_occur=	*d_B_no_solution=	*d_B_got_solution =0;
	int row = threadIdx.x;
	for (i = 0; i < COLS; i++) //calc bit_map
			{
		if (!(((*d_matrix)[row][i]) & 0x8000000000000000)) { //int debug = bit_map|(1<<((*d_matrix)[row][i]-1));int debug2 = 4|64;
			if ((bit_map | (1 << ((*d_matrix)[row][i] - 1))) == bit_map) {
				//printf("\nelement conflict, wrong branch\n");//TODO: i cannot use printf in kernel
				atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution |= 1;
				*d_conflict_pos = row * COLS + i;
				return;
			}
			bit_map |= 1 << ((*d_matrix)[row][i] - 1);
		}
	}

	for (i = 0; i < COLS; i++) {
		if (((*d_matrix)[row][i]) & 0x8000000000000000) {
			if (((~bit_map) & ((*d_matrix)[row][i])) != (*d_matrix)[row][i]) //some possibility should be cut
					{
				((*d_matrix)[row][i]) &= (~bit_map);
				//d_B_change_occur=1;
			}
			int tmp = bit_count((*d_matrix)[row][i]);
//			if (tmp == 0) {
//				*d_B_no_solution = 1;
//				d_exhaust_pos = row * COLS + i;
//				return;
//			}
			if (tmp == 1) //only one possible is left, we consider it's the right one
					{

				(*d_matrix)[row][i] = highest_bit((*d_matrix)[row][i]);
				atomicOr((int*)d_B_change_occur,1);//*d_B_change_occur |= 1;

				bit_map = 0; //update bit_map
				for (i = 0; i < COLS; i++) //recalc bit_map
						{
					if (!(((*d_matrix)[row][i]) & 0x8000000000000000)) {
						if ((bit_map | (1 << ((*d_matrix)[row][i] - 1)))
								== bit_map) {
							//printf("\nelement conflict, wrong branch\n");//TODO: i cannot use printf in kernel
							atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution |= 1;
							return;
						}
						bit_map |= 1 << ((*d_matrix)[row][i] - 1);
					}
				}

				//else return NO_SOLUTION;

			}

			if ((*d_matrix)[row][i] == 0x8000000000000000) //no possible value left, there is no solution, the puzzle is wrong
					{
				//printf("\nno value for matrix[%d][%d]\n", row, i);//TODO: i cannot use printf in kernel
				atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution |= 1;
				*d_exhaust_pos = row * COLS + i;
				return;
			}
		}
	}
	return;
}

__global__ void kernel_col_check(long long (*d_matrix)[ROWS][COLS],bool *d_B_change_occur,bool *d_B_no_solution,bool *d_B_got_solution,int *d_conflict_pos,int *d_exhaust_pos) {
	int i, bit_map = 0;

	int col = threadIdx.x;
	for (i = 0; i < ROWS; i++) //calc bit_map
			{
		if (!(((*d_matrix)[i][col]) & 0x8000000000000000)) {
			//int debug = (1<<((*d_matrix)[i][col]-1));int debug2 = bit_map|debug;
			if ((bit_map | (1 << ((*d_matrix)[i][col] - 1))) == bit_map) {
				//printf("\nelement conflict, wrong branch\n");//TODO
				atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
				*d_conflict_pos = i * COLS + col;
				return;
			}
			bit_map |= 1 << ((*d_matrix)[i][col] - 1);
		}
	}

	for (i = 0; i < ROWS; i++) {
		if (((*d_matrix)[i][col]) & 0x8000000000000000) {
			if (((~bit_map) & ((*d_matrix)[i][col])) != (*d_matrix)[i][col]) {
				((*d_matrix)[i][col]) &= (~bit_map);
				//d_B_change_occur=1;
			}

			int tmp = bit_count((*d_matrix)[i][col]);
			if (tmp == 0) {
				atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
				*d_exhaust_pos = i * COLS + col;
				return;
			}
			if (tmp == 1) //only one possible is left, we consider it's the right one
					{
				(*d_matrix)[i][col] = highest_bit((*d_matrix)[i][col]);
				atomicOr((int*)d_B_change_occur,1);//*d_B_change_occur = true;

				bit_map = 0; //update bit_map
				for (i = 0; i < ROWS; i++) //recalc bit_map
						{
					if (!(((*d_matrix)[i][col]) & 0x8000000000000000)) {
						if ((bit_map | (1 << ((*d_matrix)[i][col] - 1)))
								== bit_map) {
							//printf("\nelement conflict, wrong branch\n");//TODO
							atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
							return;
						}
						bit_map |= 1 << ((*d_matrix)[i][col] - 1);
					}
				}
			}

			if ((*d_matrix)[i][col] == 0x8000000000000000) {
				//printf("\nno value for matrix[%d][%d]\n", i, col);//TODO
				atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
				*d_exhaust_pos = i * COLS + col;
				return;
			}
		}
	}
	return;
}

__global__ void kernel_block_check(long long (*d_matrix)[ROWS][COLS],bool *d_B_change_occur,bool *d_B_no_solution,bool *d_B_got_solution,int *d_conflict_pos,int *d_exhaust_pos) {
	int i, j, bit_map = 0;

	int nth_block = threadIdx.x;
	int block_row = nth_block / ROOT;
	int block_cul = nth_block % ROOT;

	for (i = 0; i < ROOT; i++) //calc bit_map
			{
		for (j = 0; j < ROOT; j++) {
			if (!(((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]) & 0x8000000000000000)) {
				if ((bit_map
						| (1
								<< ((*d_matrix)[block_row * ROOT + i][block_cul * ROOT
										+ j] - 1))) == bit_map) {
					//printf("\nelement conflict, wrong branch\n");//TODO
					atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
					*d_conflict_pos = i * COLS + j;
					return;
				}
				bit_map |= 1
						<< ((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]
								- 1);
			}

		}

	}

	for (i = 0; i < ROOT; i++) {
		for (j = 0; j < ROOT; j++) {

			if (((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]) & 0x8000000000000000) {
				if (((~bit_map)
						& ((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]))
						!= (*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]) {
					((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]) &=
							(~bit_map);
					//d_B_change_occur=1;
				}

				int tmp = bit_count(
						(*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]);
				if (tmp == 0) {
					atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
					*d_exhaust_pos = i * COLS + j;
					return;
				}
				if (tmp == 1) //only one possible is left, we consider it's the right one
						{
					(*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j] =
							highest_bit(
									(*d_matrix)[block_row * ROOT + i][block_cul * ROOT
											+ j]);
					atomicOr((int*)d_B_change_occur,1);//*d_B_change_occur = 1;


					bit_map = 0; //update bit_map
					for (i = 0; i < ROOT; i++) //recalc bit_map
							{
						for (j = 0; j < ROOT; j++) {
							if (!(((*d_matrix)[block_row * ROOT + i][block_cul * ROOT
									+ j]) & 0x8000000000000000)) {
								if ((bit_map
										| (1
												<< ((*d_matrix)[block_row * ROOT
														+ i][block_cul * ROOT + j]
														- 1))) == bit_map) {
									//printf("\nelement conflict, wrong branch\n");//TODO
									atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
									return;
								}
								bit_map |=
										1
												<< ((*d_matrix)[block_row * ROOT
														+ i][block_cul * ROOT + j]
														- 1);
							}

						}

					}
				}

				if ((*d_matrix)[block_row * ROOT + i][block_cul * ROOT + j]
						== 0x8000000000000000) {
					//printf("\nno value for matrix[%d][%d]\n", block_row * ROOT + i,block_cul * ROOT + j);//TODO
					atomicOr((int*)d_B_no_solution,1);//*d_B_no_solution = 1;
					*d_exhaust_pos = i * COLS + j;
					return;
				}
			}

		}

	}
	return;
}

void divide_to_new(node** foo) //donot know why, reference parameter cannot be used(like &parameter), so replace with its pointer
		{
	node *p_new_node = (node*) malloc(sizeof(struct node));
	p_new_node->current_matrix = (long long (*)[ROWS][COLS]) malloc(
			ROWS * COLS * sizeof(long long));
	memcpy(p_new_node->current_matrix, (*foo)->current_matrix,
			ROWS * COLS * sizeof(long long));
	(*(p_new_node->current_matrix))[(*foo)->divide_from_x][(*foo)->divide_from_y] =
			(*foo)->current_candicate;

	p_new_node->divide_from_x = p_new_node->divide_from_y =
			p_new_node->current_candicate = 0;
	p_new_node->p_prev_node = *foo;
	simple_node_link[tree_length * 2] = (*foo)->divide_from_x * COLS
			+ (*foo)->divide_from_y;
	simple_node_link[tree_length * 2 + 1] = (*foo)->current_candicate;
	*foo = p_new_node;

	tree_length++;
	tree_change_times++;
	divide_times++;
	printf("\ntree changes %d times\n",tree_change_times);

	printf("\ntry a new branch&add a new node\nit has %d nodes now!!!\n",
			tree_length);

}

void choose_best_candicate(node ** foo) {
	int least_candicate, least_x, least_y;

	calc_least_candicate((*foo)->current_matrix, &least_x, &least_y,
			&least_candicate);

	if (least_candicate == 0xFF) {//branch exhaust
		h_B_no_solution = 1; //send a branch-over signal
		h_exhaust_pos = 0xFF;
		return;
	}
	(*foo)->divide_from_x = least_x;
	(*foo)->divide_from_y = least_y;
	(*foo)->current_candicate = highest_bit(
			(*((*foo)->current_matrix))[least_x][least_y]);
	divide_to_new(foo);

	print_simple_node_link(simple_node_link);

}

int search_solution(long long (*p_matrix)[ROWS][COLS]) {
	int i;

	h_B_change_occur = 1;
	h_B_no_solution = 0;
	int debug_count = 0;
	h_conflict_pos = h_exhaust_pos = -1;

	CUDA_CHECK_RETURN(hipMemcpy(&(*d_matrix[0][0]),&((*p_matrix)[0][0]), sizeof(long long)*ROWS*COLS,hipMemcpyHostToDevice));
//	CUDA_CHECK_RETURN(cudaMemcpyToSymbol(d_matrix, *p_matrix, sizeof(long long)*ROWS*COLS, cudaMemcpyHostToDevice));
	while (h_B_change_occur && !h_B_no_solution) //everytime we search a matrix, we must loop the three phase until no change happen
	{
		debug_count++;
		h_B_change_occur = 0; //*d_B_no_solution = 0;
		//for (i = 0; (i < ROWS) && !*d_B_no_solution; i++) {
		kernel_row_check<<<1, ROWS>>>(d_matrix,d_B_change_occur,d_B_no_solution,d_B_got_solution,d_conflict_pos,d_exhaust_pos);
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_change_occur,d_B_change_occur,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_no_solution,d_B_no_solution,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_conflict_pos,d_conflict_pos,sizeof(int),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_exhaust_pos,d_exhaust_pos,sizeof(int),hipMemcpyDeviceToHost));
		//}
		if(h_B_no_solution) break;
		//for (i = 0; (i < COLS) && !d_B_no_solution; i++) {
		kernel_col_check<<<1,ROWS>>>(d_matrix,d_B_change_occur,d_B_no_solution,d_B_got_solution,d_conflict_pos,d_exhaust_pos);
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_change_occur,d_B_change_occur,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_no_solution,d_B_no_solution,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_conflict_pos,d_conflict_pos,sizeof(int),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_exhaust_pos,d_exhaust_pos,sizeof(int),hipMemcpyDeviceToHost));
		//}
		if(h_B_no_solution) break;
		//for (i = 0; (i < ROWS) && !d_B_no_solution; i++) {
		kernel_block_check<<<1,ROWS>>>(d_matrix,d_B_change_occur,d_B_no_solution,d_B_got_solution,d_conflict_pos,d_exhaust_pos);
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_change_occur,d_B_change_occur,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_B_no_solution,d_B_no_solution,sizeof(bool),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_conflict_pos,d_conflict_pos,sizeof(int),hipMemcpyDeviceToHost));
		CUDA_CHECK_RETURN(hipMemcpy(&h_exhaust_pos,d_exhaust_pos,sizeof(int),hipMemcpyDeviceToHost));
		//}
		if(h_B_no_solution) break;
	}

	if (h_B_no_solution) {

		return NO_SOLUTION;
	}

	else {
		CUDA_CHECK_RETURN(hipMemcpy(*p_matrix, *d_matrix, sizeof(long long)*ROWS*COLS, hipMemcpyDeviceToHost));//only this result deserve copied out
		int tmp = calc_least_candicate(p_matrix, &i, &i, &i);

		if (tmp == 0xFF) {
			h_B_got_solution = 1;
			return GOT_SOLUTION;
		}
		if (tmp > 1)
			return NO_CHANGE_SO_PAUSE;
	}

}

void backward_on_tree(node **p_tree) {
	int biggest_candicate;
	free((*p_tree)->current_matrix);
	free((*p_tree));
	(*p_tree) = (*p_tree)->p_prev_node; //backward
	tree_length--;
	tree_change_times++;
	printf("\ntree changes %d times\n",tree_change_times);
	simple_node_link[tree_length * 2] = -1;
	simple_node_link[tree_length * 2 + 1] = -1;
	printf("\ndelete a wrong node,\nit has %d nodes now!!!\n", tree_length);
	print_simple_node_link(simple_node_link);

	long long (*p_matrix)[ROWS][COLS] = (*p_tree)->current_matrix;
	biggest_candicate = highest_bit(
			(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y]); //cut the highest bit
	(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y] &= (~(1
			<< (biggest_candicate - 1)));

	int debug_bit_count = bit_count(
			(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y]);
	if (bit_count(
			(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y])
			== 1) {
		(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y] =
				highest_bit(
						(*p_matrix)[(*p_tree)->divide_from_x][(*p_tree)->divide_from_y]);
		search_solution((*p_tree)->current_matrix);

	}

}

int main(void) {
	int i, tmp;
	tree_length = 0;
	tree_change_times  =0;
	memset(simple_node_link,-1,ROWS * COLS * 2*sizeof(int));

	int *debug_tree_length = &tree_length;
	int *debug_tree_change_times =&tree_change_times;
	int *debug_divide_times = &divide_times;
	int *debug_conflict_pos = &h_conflict_pos;
	int *debug_exhaust_pos = &h_exhaust_pos;
	int (*debug_simple_node_link)[ROWS * COLS * 2] = &simple_node_link;

	for (i = 0; i < ROWS * COLS; i++) {
		//scanf("%d",&tmp);
		tmp = matrix[i / COLS][i % COLS];
		matrix[i / COLS][i % COLS] = tmp ? tmp : 0xFFFFFFFFFFFFFFFF; //first bit means this is a solution, last 9 bits is the solution map;
		original_matrix[i / COLS][i % COLS] = tmp ? tmp : 0xFFFFFFFFFFFFFFFF;
	}



	//above is the c version
	//---------------------------------
	//below is the cuda version

	print_matrix(matrix);

	node head_node;
	head_node.current_matrix = &matrix;
	head_node.current_candicate = head_node.divide_from_x =
			head_node.divide_from_y = 0;

	node * tree;
	tree = &head_node;

	//choose_best_candicate(tree);
	//d_matrix = (long long(*)[ROWS][COLS])malloc(sizeof(long long*));//it is a global matrix initiated at the start, no need to alloc
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_matrix, (size_t)sizeof(long long)*ROWS*COLS));//TODO:remember to free the address
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_B_change_occur, (size_t)sizeof(bool)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_B_no_solution, (size_t)sizeof(bool)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_B_got_solution, (size_t)sizeof(bool)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_conflict_pos, (size_t)sizeof(int)));
	CUDA_CHECK_RETURN(hipMalloc((void **)&d_exhaust_pos, (size_t)sizeof(int)));

	Runtime(
		int result = search_solution(tree->current_matrix); //result only comes from search_solution
		printf("\nafter search, ");
		print_matrix(*(tree->current_matrix));
		while ((result == NO_CHANGE_SO_PAUSE) || (result == NO_SOLUTION)) {
			if (result == NO_SOLUTION) {
				if (tree == &head_node) {
					break; //game is over, totally no solution!
				}
				backward_on_tree(&tree); //call choose_best_candicate, so tree may be changed again!!!!!!
			}

			if (h_B_got_solution == 1) {
				break;
			}

			if (h_B_no_solution == 1) //branch is over, no need to search
					{
				result = NO_SOLUTION;
				continue;
			}

			choose_best_candicate(&tree); //!!!NOTE:tree has been changed to a new one!

			if (h_B_no_solution == 1) //branch is over, no need to search
					{
				result = NO_SOLUTION;
				continue;
			}

			result = search_solution(tree->current_matrix); //result only comes from search_solution
			printf("\nafter search, ");
			print_matrix(*(tree->current_matrix));
		}

	);
	if (h_B_got_solution == 1) {
		printf("\nGot a solution!\n");
		print_matrix(*(tree->current_matrix));

	}

	else {
		printf("No solution!\nThe last found matrix is like this\n");
		print_matrix(*(tree->current_matrix));
		print_candicate_num(*(tree->current_matrix));

	}

	//if(*d_matrix)//XXX
		//cudaFree(*d_matrix);
	if(d_matrix)//no need to free
		hipFree(d_matrix);

	return EXIT_SUCCESS;
}
